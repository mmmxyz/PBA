#include "hip/hip_runtime.h"

#include "utilscuda/mathfunc/mathfunc.cuh"

#include <hip/hip_runtime.h>

#include <iostream>

#include "kernel.hpp"

__constant__ fvec2* d_RestPositionList	  = nullptr;
__constant__ uint32_t* d_TriIndList	  = nullptr;
__constant__ uint32_t* d_InnerEdgeIndList = nullptr;
__constant__ uint32_t* d_EdgeList	  = nullptr;
__constant__ fvec4* d_InnerEdgeCList	  = nullptr;
__constant__ fmat2* d_AList		  = nullptr;
__constant__ float* d_VList		  = nullptr;

__constant__ fvec3* d_dx    = nullptr;
__constant__ fvec3* d_tempp = nullptr;

__constant__ float* d_LambdaContinuumList = nullptr;
__constant__ float* d_LambdaBendList	  = nullptr;
__constant__ float* d_LambdaSpringList	  = nullptr;

__constant__ uint32_t d_vertsize;
__constant__ uint32_t d_tisize;
__constant__ uint32_t d_edgesize;
__constant__ uint32_t d_InnerEdgesize;

__constant__ float d_mass;
__constant__ float d_dt;

static fvec2* cpu_d_RestPositionList	= nullptr;
static uint32_t* cpu_d_TriIndList	= nullptr;
static uint32_t* cpu_d_InnerEdgeIndList = nullptr;
static uint32_t* cpu_d_EdgeList		= nullptr;
static fvec4* cpu_d_InnerEdgeCList	= nullptr;
static fmat2* cpu_d_AList		= nullptr;
static float* cpu_d_VList		= nullptr;
static fvec3* cpu_d_dx			= nullptr;
static fvec3* cpu_d_tempp		= nullptr;
static float* cpu_d_LambdaContinuumList = nullptr;
static float* cpu_d_LambdaBendList	= nullptr;
static float* cpu_d_LambdaSpringList	= nullptr;

static fvec2* cpu_RestPositionList    = nullptr;
static uint32_t* cpu_TriIndList	      = nullptr;
static uint32_t* cpu_InnerEdgeIndList = nullptr;
static uint32_t* cpu_EdgeList	      = nullptr;
static fvec4* cpu_InnerEdgeCList      = nullptr;
static fmat2* cpu_AList		      = nullptr;
static float* cpu_VList		      = nullptr;
static float* cpu_LambdaContinuumList = nullptr;
static float* cpu_LambdaBendList      = nullptr;
static float* cpu_LambdaSpringList    = nullptr;

static uint32_t cpu_vertsize;
static uint32_t cpu_tisize;
static uint32_t cpu_edgesize;
static uint32_t cpu_InnerEdgesize;

static fvec3* cpu_dx = nullptr;

__global__ void
FemElasticProjectGPU_Kernel(const float lambda, const float mu)
{

	uint32_t i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < d_tisize / 3) {
		fvec3 x0 = d_tempp[d_TriIndList[3 * i + 0]];
		fvec3 x1 = d_tempp[d_TriIndList[3 * i + 1]];
		fvec3 x2 = d_tempp[d_TriIndList[3 * i + 2]];

		fmat2 A	 = d_AList[i];
		fmat32 F = fmat32(x1 - x0, x2 - x0) * A;

		float C00 = F.m[2 * 0 + 0] * F.m[2 * 0 + 0] + F.m[2 * 1 + 0] * F.m[2 * 1 + 0] + F.m[2 * 2 + 0] * F.m[2 * 2 + 0];
		float C01 = F.m[2 * 0 + 0] * F.m[2 * 0 + 1] + F.m[2 * 1 + 0] * F.m[2 * 1 + 1] + F.m[2 * 2 + 0] * F.m[2 * 2 + 1];
		float C11 = F.m[2 * 0 + 1] * F.m[2 * 0 + 1] + F.m[2 * 1 + 1] * F.m[2 * 1 + 1] + F.m[2 * 2 + 1] * F.m[2 * 2 + 1];

		fmat2 E;
		E.m[0] = 0.5f * (C00 - 1);
		E.m[1] = 0.5f * C01;
		E.m[2] = 0.5f * C01;
		E.m[3] = 0.5f * (C11 - 1);

		float V = d_VList[i];
		V *= 0.01f;

		float W	 = V * (mu * E.sqlength() + 0.5f * lambda * E.trace() * E.trace());
		fmat32 B = V * (2.0f * mu * F * E + lambda * E.trace() * F);

		//printf("%f\n", C00);
		//printf("%f\n", C01);
		//printf("%f\n", C11);
		//if (i == 0) {
		//	printf("%f ", x0.x);
		//	printf("%f ", x0.y);
		//	printf("%f   ", x0.z);

		//	printf("%f ", X0.x);
		//	printf("%f \n", X0.y);
		//}

		if (W > 0.0) {
			float C = std::sqrt(2.0 * W);

			fmat32 BAt = B * A.transpose();

			fvec3 dC1 = (1.0f / C) * fvec3(BAt.m[0], BAt.m[2], BAt.m[4]);
			fvec3 dC2 = (1.0f / C) * fvec3(BAt.m[1], BAt.m[3], BAt.m[5]);
			fvec3 dC0 = -(dC1 + dC2);

			float dtdtdlambda = (-C - d_LambdaContinuumList[i]) / ((dC0.sqlength() + dC1.sqlength() + dC2.sqlength()) / d_mass + 1.0 / (d_dt * d_dt));
			dtdtdlambda *= 0.4;

			d_dx[3 * i + 0] = dtdtdlambda * (1.0f / d_mass) * dC0;
			d_dx[3 * i + 1] = dtdtdlambda * (1.0f / d_mass) * dC1;
			d_dx[3 * i + 2] = dtdtdlambda * (1.0f / d_mass) * dC2;

			d_LambdaContinuumList[i] += dtdtdlambda / (d_dt * d_dt);
		} else {

			d_dx[3 * i + 0] = fvec3(0.0);
			d_dx[3 * i + 1] = fvec3(0.0);
			d_dx[3 * i + 2] = fvec3(0.0);
		}
	}
}

__global__ void
FemBendProjectGPU_Kernel(const float bendCof)
{

	uint32_t i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < d_InnerEdgesize / 4) {

		fvec3 x0 = d_tempp[d_InnerEdgeIndList[4 * i + 0]];
		fvec3 x1 = d_tempp[d_InnerEdgeIndList[4 * i + 1]];
		fvec3 x2 = d_tempp[d_InnerEdgeIndList[4 * i + 2]];
		fvec3 x3 = d_tempp[d_InnerEdgeIndList[4 * i + 3]];

		fvec4 Cot = d_InnerEdgeCList[i];

		fmat4 X	   = fmat4(fvec4(x0), fvec4(x1), fvec4(x2), fvec4(x3));
		fvec4 XCot = X * Cot;

		float Q = bendCof * XCot.sqlength();

		if (Q > 0.0) {
			float C = std::sqrt(2.0 * Q);

			fvec3 dC0 = (1.0f / C) * bendCof * fvec3(XCot.x * Cot.x, XCot.y * Cot.x, XCot.z * Cot.x);
			fvec3 dC1 = (1.0f / C) * bendCof * fvec3(XCot.x * Cot.y, XCot.y * Cot.y, XCot.z * Cot.y);
			fvec3 dC2 = (1.0f / C) * bendCof * fvec3(XCot.x * Cot.z, XCot.y * Cot.z, XCot.z * Cot.z);
			fvec3 dC3 = (1.0f / C) * bendCof * fvec3(XCot.x * Cot.w, XCot.y * Cot.w, XCot.z * Cot.w);

			float dtdtdlambda = (-C - d_LambdaBendList[i]) / ((dC0.sqlength() + dC1.sqlength() + dC2.sqlength() + dC3.sqlength()) / d_mass + 1.0 / (d_dt * d_dt));
			dtdtdlambda *= 0.8;

			d_dx[4 * i + 0] = dtdtdlambda * (1.0f / d_mass) * dC0;
			d_dx[4 * i + 1] = dtdtdlambda * (1.0f / d_mass) * dC1;
			d_dx[4 * i + 2] = dtdtdlambda * (1.0f / d_mass) * dC2;
			d_dx[4 * i + 3] = dtdtdlambda * (1.0f / d_mass) * dC3;

			d_LambdaBendList[i] += dtdtdlambda / (d_dt * d_dt);
		} else {
			d_dx[4 * i + 0] = fvec3(0.0);
			d_dx[4 * i + 1] = fvec3(0.0);
			d_dx[4 * i + 2] = fvec3(0.0);
			d_dx[4 * i + 3] = fvec3(0.0);
		}
	}
}

__global__ void
FemAreaProjectGPU_Kernel()
{

	uint32_t i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < d_InnerEdgesize / 4) {
		fvec3 x0 = d_tempp[d_TriIndList[3 * i + 0]];
		fvec3 x1 = d_tempp[d_TriIndList[3 * i + 1]];
		fvec3 x2 = d_tempp[d_TriIndList[3 * i + 2]];

		float V = d_VList[i];

		float C = ((x1 - x0).cross(x2 - x0)).sqlength() - 4.0 * V * V;

		if (abs(C) > 0.001) {

			fvec3 dC1 = 2.0f * (x2 - x0).cross((x1 - x0).cross(x2 - x0));
			fvec3 dC2 = 2.0f * (x1 - x0).cross((x2 - x0).cross(x1 - x0));
			fvec3 dC0 = -(dC1 + dC2);

			float dtdtdlambda = (-C) / ((dC0.sqlength() + dC1.sqlength() + dC2.sqlength()) / d_mass);

			d_dx[3 * i + 0] = dtdtdlambda * (1.0f / d_mass) * dC0;
			d_dx[3 * i + 1] = dtdtdlambda * (1.0f / d_mass) * dC1;
			d_dx[3 * i + 2] = dtdtdlambda * (1.0f / d_mass) * dC2;

		} else {

			d_dx[3 * i + 0] = fvec3(0.0);
			d_dx[3 * i + 1] = fvec3(0.0);
			d_dx[3 * i + 2] = fvec3(0.0);
		}
	}
}

__global__ void
updatetempp_Kernel(fvec3* const tempp, const fvec3* const dx, const uint32_t* const TriIndList, const uint32_t N)
{
	uint32_t i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < 3 * 2 * (N - 1) * (N - 1)) {
		tempp[TriIndList[i]] = tempp[TriIndList[i]] + dx[i];
	}
}

__global__ void
ClearLambda()
{
	uint32_t i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < d_tisize / 3) {
		d_LambdaContinuumList[i] = 0.0;
	}
	if (i < d_edgesize / 2) {
		d_LambdaSpringList[i] = 0.0;
	}
	if (i < d_InnerEdgesize / 4) {
		d_LambdaBendList[i] = 0.0;
	}
}

__global__ void
PrintInfo()
{
	printf("vertsize %d \n", d_vertsize);
	printf("tisize %d \n", d_tisize);
	printf("edgesize %d \n", d_edgesize);
	printf("InnerEdgesize %d \n", d_InnerEdgesize);

	for (int i = 0; i < d_InnerEdgesize / 4; i++) {
		printf("%f  ", d_InnerEdgeCList[i].x);
		printf("%f  ", d_InnerEdgeCList[i].y);
		printf("%f  ", d_InnerEdgeCList[i].z);
		printf("%f  ", d_InnerEdgeCList[i].w);
		printf("%d ", d_InnerEdgeIndList[4 * i + 0]);
		printf("%d ", d_InnerEdgeIndList[4 * i + 1]);
		printf("%d ", d_InnerEdgeIndList[4 * i + 2]);
		printf("%d \n", d_InnerEdgeIndList[4 * i + 3]);
	}
}

void Init(MeshInfo& mInfo)
{

	cpu_RestPositionList = mInfo.Restvertdata;
	cpu_TriIndList	     = mInfo.tilist;
	cpu_InnerEdgeIndList = mInfo.InnerEdgelist;
	cpu_EdgeList	     = mInfo.edgelist;
	cpu_InnerEdgeCList   = mInfo.InnerEdgeClist;
	cpu_AList	     = mInfo.Alist;
	cpu_VList	     = mInfo.Vlist;

	cpu_vertsize	  = mInfo.vertsize;
	cpu_tisize	  = mInfo.tisize;
	cpu_edgesize	  = mInfo.edgesize;
	cpu_InnerEdgesize = mInfo.InnerEdgesize;

	//memory allocation
	hipMalloc(&cpu_d_RestPositionList, cpu_vertsize * sizeof(fvec2));
	hipMalloc(&cpu_d_TriIndList, cpu_tisize * sizeof(uint32_t));
	hipMalloc(&cpu_d_InnerEdgeIndList, cpu_InnerEdgesize * sizeof(uint32_t));
	hipMalloc(&cpu_d_InnerEdgeCList, (cpu_InnerEdgesize / 4) * sizeof(fvec4));
	hipMalloc(&cpu_d_EdgeList, cpu_edgesize * sizeof(uint32_t));
	hipMalloc(&cpu_d_AList, (cpu_tisize / 3) * sizeof(fmat2));
	hipMalloc(&cpu_d_VList, (cpu_tisize / 3) * sizeof(float));
	hipMalloc(&cpu_d_dx, max(max(cpu_tisize, cpu_edgesize), cpu_InnerEdgesize) * sizeof(fvec3));
	hipMalloc(&cpu_d_tempp, cpu_vertsize * sizeof(fvec3));

	hipMalloc(&cpu_d_LambdaContinuumList, (cpu_tisize / 3) * sizeof(float));
	hipMalloc(&cpu_d_LambdaBendList, (cpu_InnerEdgesize / 4) * sizeof(float));
	hipMalloc(&cpu_d_LambdaSpringList, (cpu_edgesize / 2) * sizeof(float));

	hipDeviceSynchronize();

	//move pointer

	hipMemcpyToSymbol(HIP_SYMBOL(d_RestPositionList), &cpu_d_RestPositionList, sizeof(fvec2*));
	hipMemcpyToSymbol(HIP_SYMBOL(d_TriIndList), &cpu_d_TriIndList, sizeof(uint32_t*));
	hipMemcpyToSymbol(HIP_SYMBOL(d_InnerEdgeIndList), &cpu_d_InnerEdgeIndList, sizeof(uint32_t*));
	hipMemcpyToSymbol(HIP_SYMBOL(d_InnerEdgeCList), &cpu_d_InnerEdgeCList, sizeof(fvec4*));
	hipMemcpyToSymbol(HIP_SYMBOL(d_EdgeList), &cpu_d_EdgeList, sizeof(uint32_t*));
	hipMemcpyToSymbol(HIP_SYMBOL(d_AList), &cpu_d_AList, sizeof(fmat2*));
	hipMemcpyToSymbol(HIP_SYMBOL(d_VList), &cpu_d_VList, sizeof(float*));

	hipMemcpyToSymbol(HIP_SYMBOL(d_dx), &cpu_d_dx, sizeof(fvec3*));
	hipMemcpyToSymbol(HIP_SYMBOL(d_tempp), &cpu_d_tempp, sizeof(fvec3*));

	hipMemcpyToSymbol(HIP_SYMBOL(d_LambdaContinuumList), &cpu_d_LambdaContinuumList, sizeof(float*));
	hipMemcpyToSymbol(HIP_SYMBOL(d_LambdaBendList), &cpu_d_LambdaBendList, sizeof(float*));
	hipMemcpyToSymbol(HIP_SYMBOL(d_LambdaSpringList), &cpu_d_LambdaSpringList, sizeof(float*));

	hipMemcpyToSymbol(HIP_SYMBOL(d_vertsize), &cpu_vertsize, sizeof(uint32_t));
	hipMemcpyToSymbol(HIP_SYMBOL(d_tisize), &cpu_tisize, sizeof(uint32_t));
	hipMemcpyToSymbol(HIP_SYMBOL(d_edgesize), &cpu_edgesize, sizeof(uint32_t));
	hipMemcpyToSymbol(HIP_SYMBOL(d_InnerEdgesize), &cpu_InnerEdgesize, sizeof(uint32_t));

	hipMemcpyToSymbol(HIP_SYMBOL(d_mass), &mInfo.mass, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(d_dt), &mInfo.dt, sizeof(float));

	//move static information
	hipMemcpy(cpu_d_RestPositionList, cpu_RestPositionList, cpu_vertsize * sizeof(fvec2), hipMemcpyHostToDevice);
	hipMemcpy(cpu_d_TriIndList, cpu_TriIndList, cpu_tisize * sizeof(uint32_t), hipMemcpyHostToDevice);
	hipMemcpy(cpu_d_InnerEdgeIndList, cpu_InnerEdgeIndList, cpu_InnerEdgesize * sizeof(uint32_t), hipMemcpyHostToDevice);
	hipMemcpy(cpu_d_InnerEdgeCList, cpu_InnerEdgeCList, (cpu_InnerEdgesize / 4) * sizeof(fvec4), hipMemcpyHostToDevice);
	hipMemcpy(cpu_d_EdgeList, cpu_EdgeList, cpu_edgesize * sizeof(uint32_t), hipMemcpyHostToDevice);
	hipMemcpy(cpu_d_AList, cpu_AList, (cpu_tisize / 3) * sizeof(fmat2), hipMemcpyHostToDevice);
	hipMemcpy(cpu_d_VList, cpu_VList, (cpu_tisize / 3) * sizeof(float), hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	cpu_dx = new fvec3[max(max(cpu_tisize, cpu_edgesize), cpu_InnerEdgesize)];

	//printf("vertsize %d \n", cpu_vertsize);
	//printf("tisize %d \n", cpu_tisize);
	//printf("edgesize %d \n", cpu_edgesize);
	//printf("InnerEdgesize %d \n", cpu_InnerEdgesize);

	//PrintInfo<<<1, 1>>>();

	//for (int i = 0; i < cpu_InnerEdgesize / 4; i++) {
	//	printf("%f  ", cpu_InnerEdgeCList[i].x);
	//	printf("%f  ", cpu_InnerEdgeCList[i].y);
	//	printf("%f  ", cpu_InnerEdgeCList[i].z);
	//	printf("%f  ", cpu_InnerEdgeCList[i].w);
	//	printf("%d ", cpu_InnerEdgeIndList[4 * i + 0]);
	//	printf("%d ", cpu_InnerEdgeIndList[4 * i + 1]);
	//	printf("%d ", cpu_InnerEdgeIndList[4 * i + 2]);
	//	printf("%d \n", cpu_InnerEdgeIndList[4 * i + 3]);
	//}
}

void ClearLambdaGPU()
{
	ClearLambda<<<max(max(cpu_tisize / 3, cpu_edgesize / 2), cpu_InnerEdgesize / 4) / 32 + 1, 32>>>();
}

void FemBendProjectGPU(fvec3* const tempp, const float bendCof)
{

	hipMemcpy(cpu_d_tempp, tempp, cpu_vertsize * sizeof(fvec3), hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	uint32_t IEhsize = cpu_InnerEdgesize / 4;
	uint32_t Ds	 = IEhsize / 320;
	FemBendProjectGPU_Kernel<<<Ds + 1, 320>>>(bendCof);
	hipDeviceSynchronize();

	hipMemcpy(cpu_dx, cpu_d_dx, cpu_InnerEdgesize * sizeof(fvec3), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	for (uint32_t i = 0; i < cpu_InnerEdgesize; i++) {
		tempp[cpu_InnerEdgeIndList[i]] = tempp[cpu_InnerEdgeIndList[i]] + cpu_dx[i];
	}
}

void FemAreaProjectGPU(fvec3* const tempp)
{
	hipMemcpy(cpu_d_tempp, tempp, cpu_vertsize * sizeof(fvec3), hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	uint32_t Thsize = cpu_tisize / 3;
	uint32_t Ds	= Thsize / 320;
	FemAreaProjectGPU_Kernel<<<Ds + 1, 320>>>();
	hipDeviceSynchronize();

	hipMemcpy(cpu_dx, cpu_d_dx, cpu_tisize * sizeof(fvec3), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	for (uint32_t i = 0; i < cpu_tisize; i++) {
		tempp[cpu_TriIndList[i]] = tempp[cpu_TriIndList[i]] + cpu_dx[i];
	}
}

void FemElasticProjectGPU(fvec3* const tempp, const float lambda, const float mu)
{
	hipMemcpy(cpu_d_tempp, tempp, cpu_vertsize * sizeof(fvec3), hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	uint32_t Thsize = cpu_tisize / 3;
	uint32_t Ds	= Thsize / 320;
	FemElasticProjectGPU_Kernel<<<Ds + 1, 320>>>(lambda, mu);
	hipDeviceSynchronize();

	hipMemcpy(cpu_dx, cpu_d_dx, cpu_tisize * sizeof(fvec3), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	for (uint32_t i = 0; i < cpu_tisize; i++) {
		tempp[cpu_TriIndList[i]] = tempp[cpu_TriIndList[i]] + cpu_dx[i];
	}
}
